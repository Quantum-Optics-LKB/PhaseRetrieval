#include "hip/hip_runtime.h"
//Kernels for the WISH routines
// Compiler command to test compile using nvcc :
// nvcc -gencode=arch=compute_61,code=sm_61 --use_fast_math -I=/home/tangui/anaconda3/lib/python3.8/site-packages/cupy/_core/include,/home/tangui/anaconda3/lib/python3.8/site-packages/cupy/_core/include/cupy/_cuda/cuda-11.4,/usr/local/cuda/include -ftz=true --cubin -o=kernels kernels.cu
//block = (8, 512, 512)
//grid = (SLM.shape[0]//block[0], SLM.shape[1]//block[1], SLM.shape[2]//block[2])
#include <cupy/complex.cuh>
#include <hipfft/hipfft.h>
#include <stdio.h>

extern "C"{
  __global__ void impose_amp(const complex<float>* y, const complex<float>* x, complex<float>* out, const int N0, const int N1, const int N2){

    int i0 = blockDim.x * blockIdx.x + threadIdx.x;
    int i1 = blockDim.y * blockIdx.y + threadIdx.y;
    int i2 = blockDim.z * blockIdx.z + threadIdx.z;
    int tid = N1*N2*i0 + N2*i1 + i2;
    // //prevent illegal memory access
    if (i0 < N0 && i1 < N1 && i2 < N2){
        
    float ang = arg(x[tid]);
    out[tid] = abs(y[tid]) * exp(complex<float>(0.0f, 1.0f) * ang);
    
    }
  }

  __global__ void multiply_conjugate(const complex<float>* y, complex<float>* x, const int N0, const int N1, const int N2){

    // int tid =  blockDim.x * blockIdx.x + threadIdx.x;
    int i0 = blockDim.x * blockIdx.x + threadIdx.x;
    int i1 = blockDim.y * blockIdx.y + threadIdx.y;
    int i2 = blockDim.z * blockIdx.z + threadIdx.z;
    int tid = N1*N2*i0 + N2*i1 + i2;
    
    if (i0 < N0 && i1 < N1 && i2 < N2){
    
      x[tid] *= conj(y[tid]);
    
    }
  }
  //pass .handle attr of cupy fft_plan (int ptr)
  // pass A0.data.ptr data pointer from python side
  // __global__ void frt_gpu_vec_s(thrust::device_vector<thrust::complex<float>>& A0, const float d1x, const float d1y,
  //   const float wv, const float z, const hipfftHandle plan){

  //   // int tid = blockDim.x * blockIdx.x + threadIdx.x;
  //   // auto _A0 = const_cast<hipfftComplex*>(reinterpret_cast<const hipfftComplex*>(thrust::raw_pointer_cast(A0.data())));
  //   // if (z>0){
  //   //   hipfftExecC2C(plan, _A0, _A0, HIPFFT_FORWARD);
  //   //   A0[tid].x *= d1x * d1y;
  //   //   A0[tid].y *= d1x * d1y;
  //   // }
  //   // else{
  //   //   hipfftExecC2C(plan, _A0, _A0, HIPFFT_BACKWARD);
  //   //   A0[tid].x *= d1x * d1y;
  //   //   A0[tid].y *= d1x * d1y;
  //   // }
    
  //   // A0[tid] = A0[tid] / (complex<float>(0.0f, 1.0f) * wv * z);
  //   printf(typeid(A0).name());

  // }
}
