#include "hip/hip_runtime.h"
//Kernels for the WISH routines
// Compiler command to test compile using nvcc :
// nvcc -gencode=arch=compute_61,code=sm_61 --use_fast_math -I=/home/tangui/anaconda3/lib/python3.8/site-packages/cupy/_core/include,/home/tangui/anaconda3/lib/python3.8/site-packages/cupy/_core/include/cupy/_cuda/cuda-11.3,/usr/local/cuda/include -ftz=true --cubin -o=kernels kernels.cu
//block = (8, 512, 512)
//grid = (SLM.shape[0]//block[0], SLM.shape[1]//block[1], SLM.shape[2]//block[2])
#include <cupy/complex.cuh>
#include <hipfft/hipfft.h>
#include <stdio.h>

extern "C"{
  __global__ void impose_amp(const complex<float>* y, complex<float>* x, const int Nx, const int Ny, const int Nz){

    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iy = blockDim.y * blockIdx.y + threadIdx.y;
    int iz = blockDim.z * blockIdx.z + threadIdx.z;
    int tid = Ny*Nz*ix+ Nz*iy+ iz;
    // h*d*(8*bx + tx)+ d*(8*by + ty)+ (8*bz + tz)
    // //prevent illegal memory access
    if (tid < Nx*Ny*Nz){

    x[tid] = abs(y[tid]) * exp(complex<float>(0.0f, 1.0f) * arg(x[tid]));
    
    }
  }

  __global__ void multiply_conjugate(const complex<float>* y, complex<float>* x, const int Nx, const int Ny, const int Nz){

    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iy = blockDim.y * blockIdx.y + threadIdx.y;
    int iz = blockDim.z * blockIdx.z + threadIdx.z;
    int tid = Ny*Nz*ix+ Nz*iy+ iz;
    
    if (tid < Nx*Ny*Nz){
    
      x[tid] *= conj(y[tid]);
    
    }
  }
  //pass .handle attr of cupy fft_plan (int ptr)
  // pass A0.data.ptr data pointer from python side
  // __global__ void frt_gpu_vec_s(void *A0, const float d1x, const float d1y,
  //   const float wv, const float z, const hipfftHandle plan){

  //   // int tid = blockDim.x * blockIdx.x + threadIdx.x;
  //   // hipComplex* _A0 = (hipComplex*)thrust::raw_pointer_cast(A0.data());
  //   // if (z>0){
  //   //   hipfftExecC2C(plan, _A0, _A0, HIPFFT_FORWARD);
  //   //   A0[tid].x *= d1x * d1y;
  //   //   A0[tid].y *= d1x * d1y;
  //   // }
  //   // else{
  //   //   hipfftExecC2C(plan, _A0, _A0, HIPFFT_BACKWARD);
  //   //   A0[tid].x *= d1x * d1y;
  //   //   A0[tid].y *= d1x * d1y;
  //   // }
    
  //   // A0[tid] = A0[tid] / (complex<float>(0.0f, 1.0f) * wv * z);
  //   printf(typeid(A0).name());

  // }
}
